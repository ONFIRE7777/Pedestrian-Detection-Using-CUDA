#include "hip/hip_runtime.h"
/*
  Host main routine
*/
#include "headers.h"

using namespace cv;
using namespace std;


int main(void)
{
    bool DEBUG = true;
    printf("Inside Host Code\n");
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    /*
        Kernel Loading 
    */

/*  
    int filterH, filterW;
    cout<<"Enter filter height and width:"<<endl;
    cin >> filterH >> filterW;
    
    // Print the vector length to be used, and compute its size
    int filterSize = filterH * filterW;
    size_t filterSizeInByte = filterSize * sizeof(float);

    // Allocate memory for filter kernel
    float *h_filterKernel = (float *)malloc(filterSizeInByte);

    // Initialize the host input vectors
    for (int i = 0; i < filterSize; ++i)
        cin>> h_filterKernel[i];

    // Transfer host data to constant device memory
    hipMemcpyToSymbol(HIP_SYMBOL( d_filterKernel), h_filterKernel, filterSizeInByte, 0,hipMemcpyHostToDevice);
*/
    /*
        Image Loading 
    */

    // OpenCV code for reading image
    Mat img = imread("../persons/person_024.bmp",1);
   
    imshow("PersonImage",img);
    waitKey(0);
   
    // Padding required depending on kernel size
    int padding = 2;

    // Providing padding to image
    int paddedR = img.rows + padding;
    int paddedC = img.cols + padding;

    size_t imageSize = paddedR * paddedC * sizeof(float);

    // Allocate memory for Blue Channel of image
    float *h_B = (float *)malloc(imageSize);

    // Allocate memory for Green Channel of image
    float *h_G = (float *)malloc(imageSize);

    // Allocate memory for Red Channel of image
    float *h_R = (float *)malloc(imageSize);

    // Verify that allocations succeeded
    if (h_B == NULL || h_G == NULL || h_R == NULL)
    {
        fprintf(stderr, "Failed to allocate memory for image!\n");
        exit(EXIT_FAILURE);
    }

    // Converting Mat to 1D array 
    for (int i = 0; i < paddedR; ++i)
        for (int j = 0; j < paddedC; ++j)
        {
            if( i==0 || i==paddedR-1 || j==0 || j==paddedC-1 )
            {
                h_B[i*paddedC + j] = 0;
                h_G[i*paddedC + j] = 0;
                h_R[i*paddedC + j] = 0;
            }                
            else
            {
                h_B[i*paddedC + j] = img.at<Vec3b>(i,j)[0];
                h_G[i*paddedC + j] = img.at<Vec3b>(i,j)[1];
                h_R[i*paddedC + j] = img.at<Vec3b>(i,j)[2];
            }
        }


    // Verify that the channel array is correct
    if(DEBUG)
    {   
        Mat checkImage(paddedR,paddedC, CV_8UC1, Scalar(0));
        for (int i = 0; i < paddedR*paddedC; ++i)
        {
            checkImage.at<uchar>(i/paddedC,i%paddedC) = h_B[i];
        }
        imshow("checkImage", checkImage);
        waitKey(0);
    }

    
    // Allocate the device memory for Blue Channel
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for B channel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device memory for Green Channel
    float *d_G = NULL;
    err = hipMalloc((void **)&d_G, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device memory for Red Channel
    float *d_R = NULL;
    err = hipMalloc((void **)&d_R, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //  Copy image from host memory to device memory 
    printf("Copying image from host memory to device memory.\n");
    err = hipMemcpy(d_B, h_B, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy blue channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_G, h_G, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy green channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_R, h_R, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy red channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    

    

    // Allocate the device memory for output
    float *d_output = NULL;
    // err = hipMalloc((void **)&d_output, tileSize);
    err = hipMalloc((void **)&d_output, img.rows*img.cols*sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate memory output gradient values
    // float *h_output = (float *)malloc(tileSize);
    float *h_output = (float *)malloc(img.rows*img.cols*sizeof(float));

    // Size of image block that will have its gradient calc. in one kernel call
    int blockX = 30, blockY = 30;

    // Size to be allocated for shared memory inside kernel
    int tileX = blockX+padding;
    int tileY = blockY+padding; 
    size_t tileSize = (tileX)*(tileY)*sizeof(float);
    // Specifying execution configuration
    // cout<<"Verification: "<<ceil(img.rows/blockX)<<" "<<(img.cols/blockY)<<endl;
    dim3 X(ceil(img.rows/blockX),ceil(img.cols/blockY));
    // dim3 X(1,1);
    dim3 Y(tileX/4  ,tileY);

    convolution<<<X, Y, tileSize>>>(d_B, paddedR, paddedC, blockX, blockY, d_output);
   
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch convolution kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output, d_output, tileSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    Mat featureImage(img.rows, img.cols, CV_8UC1, Scalar(0));
    // Verify that the result vector is correct
    for (int i = 0; i < img.rows*img.cols; ++i)
    {
        featureImage.at<uchar>(i/img.cols,i%img.cols) = h_output[i];
        // cout<<(int)featureImage.at<uchar>(i/tileY,i%tileY)<<endl;
    }
    // imshow("Input Image", origImage);
    imshow("Output Image", featureImage);
    waitKey(0);

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_output);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_G);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_R);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array R (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_B);
    free(h_G);
    free(h_R);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

