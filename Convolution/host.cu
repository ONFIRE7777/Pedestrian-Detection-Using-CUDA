#include "hip/hip_runtime.h"
/*  Host main routine   */
#include "headers.h"

using namespace cv;
using namespace std;


int main(void)
{
    bool DEBUG = true;
    printf("Inside Host Code\n");
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    /*   Image Loading   */

    // OpenCV code for reading image
    Mat img = imread("../persons/person_024.bmp",1);

    // To verify if original image is loaded properly 
    if(DEBUG)
    {        
        imshow("PersonImage",img);
        waitKey(0);
    }     
   
    // Padding required depending on kernel size
    // here kernel size is fixed always as 1 so 2*1
    int padding = 2;

    // Providing padding to image
    // X will be treated as rows and Y as cols
    int paddedX = img.rows + padding;
    int paddedY = img.cols + padding;

    size_t imageSize = img.rows * img.cols * sizeof(float);
    size_t paddedImageSize = paddedX * paddedY * sizeof(float);

    // Allocate memory for Blue Channel of image
    float *h_B = (float *)malloc(paddedImageSize);

    // Allocate memory for Green Channel of image
    float *h_G = (float *)malloc(paddedImageSize);

    // Allocate memory for Red Channel of image
    float *h_R = (float *)malloc(paddedImageSize);

    // Verify that allocations succeeded
    if (h_B == NULL || h_G == NULL || h_R == NULL)
    {
        fprintf(stderr, "Failed to allocate memory for image!\n");
        exit(EXIT_FAILURE);
    }

    // Converting Mat to 1D array 
    for (int i = 0; i < paddedX; ++i)
        for (int j = 0; j < paddedY; ++j)
        {
            if( i==0 || i==paddedX-1 || j==0 || j==paddedY-1 )
            {
                h_B[i*paddedY + j] = 0;
                h_G[i*paddedY + j] = 0;
                h_R[i*paddedY + j] = 0;
            }                
            else
            {
                h_B[i*paddedY + j] = img.at<Vec3b>(i,j)[0];
                h_G[i*paddedY + j] = img.at<Vec3b>(i,j)[1];
                h_R[i*paddedY + j] = img.at<Vec3b>(i,j)[2];
            }
        }


    // Verify that the channel array is correct
    if(DEBUG)
    {   
        Mat checkImage(paddedX,paddedY, CV_8UC1, Scalar(0));
        for (int i = 0; i < paddedX*paddedY; ++i)
        {
            checkImage.at<uchar>(i/paddedY,i%paddedY) = h_B[i];
        }
        imshow("checkImage", checkImage);
        waitKey(0);
    }

    
    // Allocate the device memory for Blue Channel
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, paddedImageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for B channel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device memory for Green Channel
    float *d_G = NULL;
    err = hipMalloc((void **)&d_G, paddedImageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device memory for Red Channel
    float *d_R = NULL;
    err = hipMalloc((void **)&d_R, paddedImageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //  Copy image from host memory to device memory 
    printf("Copying image from host memory to device memory.\n");
    err = hipMemcpy(d_B, h_B, paddedImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy blue channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_G, h_G, paddedImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy green channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_R, h_R, paddedImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy red channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory for output gradient values: magnitude and angle
    float *h_outputMag = (float *)malloc(imageSize);
    float *h_outputAng = (float *)malloc(imageSize);

    // Allocate the device memory for output
    float *d_outputMag = NULL;
    err = hipMalloc((void **)&d_outputMag, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_outputAng = NULL;
    err = hipMalloc((void **)&d_outputAng, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Image is divided in no of image blocks gradients for each will be calculated parallely
    // Size of image block that will have its gradient calc. in one kernel call
    int blockX = 32, blockY = 32;

    // Size to be allocated for shared memory inside kernel
    // This is the size of block along with padding so that convolution can be done 
    // at the border points
    int tileX = blockX + padding;
    int tileY = blockY + padding; 
    size_t tileSize = (tileX)*(tileY)*sizeof(float);

    // for each tile only 4th the threads are allocated and then reused accordingly
    int blockDimX = ceil((double)tileX/4), blockDimY = tileY;

    // the no. of thread blocks that have to be launched will be the no. of image rows and cols 
    // divided by the no. of pixel we wish to keep in one block of image
    int gridDimX = ceil((double)img.rows/blockX), gridDimY = ceil((double)img.cols/blockY);

    // Specifying execution configuration
    dim3 X(gridDimX,gridDimY);
    dim3 Y(blockDimX,blockDimY);
    convolution<<<X, Y, tileSize>>>(d_B, paddedX, paddedY, blockX, blockY, d_outputMag, d_outputAng, img.rows, img.cols);
   
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch convolution kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_outputMag, d_outputMag, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_outputAng, d_outputAng, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the resulting image is correct
    Mat magImage(img.rows, img.cols, CV_8UC1, Scalar(0));
    Mat angleImage(img.rows, img.cols, CV_8UC1, Scalar(0));
    for (int i = 0; i < img.rows*img.cols; ++i)
    {
        magImage.at<uchar>(i/img.cols,i%img.cols) = h_outputMag[i];
        angleImage.at<uchar>(i/img.cols,i%img.cols) = h_outputAng[i];
    }
    imshow("Output Angle", angleImage);
    imshow("Output Maginitude", magImage);
    waitKey(0);
   


    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_outputMag);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_outputAng);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_G);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_R);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array R (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_B);
    free(h_G);
    free(h_R);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

