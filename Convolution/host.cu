#include "hip/hip_runtime.h"
/*
  Host main routine
*/
#include "headers.h"

using namespace cv;
using namespace std;


int main(void)
{
    bool DEBUG = true;
    printf("Inside Host Code\n");
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    /*
        Image Loading 
    */

    // OpenCV code for reading image
    Mat img = imread("../persons/person_024.bmp",1);

    /* 
        // To verify if original image is loaded properly 
        imshow("PersonImage",img);
        waitKey(0);
       
    */    
   
    // Padding required depending on kernel size
    int padding = 2;

    // Providing padding to image X-rows and Y-cols
    int paddedX = img.rows + padding;
    int paddedY = img.cols + padding;

    size_t imageSize = img.rows * img.cols * sizeof(float);
    size_t paddedImageSize = paddedX * paddedY * sizeof(float);

    // Allocate memory for Blue Channel of image
    float *h_B = (float *)malloc(paddedImageSize);

    // Allocate memory for Green Channel of image
    float *h_G = (float *)malloc(paddedImageSize);

    // Allocate memory for Red Channel of image
    float *h_R = (float *)malloc(paddedImageSize);

    // Verify that allocations succeeded
    if (h_B == NULL || h_G == NULL || h_R == NULL)
    {
        fprintf(stderr, "Failed to allocate memory for image!\n");
        exit(EXIT_FAILURE);
    }

    // Converting Mat to 1D array 
    for (int i = 0; i < paddedX; ++i)
        for (int j = 0; j < paddedY; ++j)
        {
            if( i==0 || i==paddedX-1 || j==0 || j==paddedY-1 )
            {
                h_B[i*paddedY + j] = 0;
                h_G[i*paddedY + j] = 0;
                h_R[i*paddedY + j] = 0;
            }                
            else
            {
                h_B[i*paddedY + j] = img.at<Vec3b>(i,j)[0];
                h_G[i*paddedY + j] = img.at<Vec3b>(i,j)[1];
                h_R[i*paddedY + j] = img.at<Vec3b>(i,j)[2];
            }
        }


    // Verify that the channel array is correct
    if(DEBUG)
    {   
        Mat checkImage(paddedX,paddedY, CV_8UC1, Scalar(0));
        for (int i = 0; i < paddedX*paddedY; ++i)
        {
            checkImage.at<uchar>(i/paddedY,i%paddedY) = h_B[i];
        }
        imshow("checkImage", checkImage);
        waitKey(0);
    }

    
    // Allocate the device memory for Blue Channel
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, paddedImageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for B channel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device memory for Green Channel
    float *d_G = NULL;
    err = hipMalloc((void **)&d_G, paddedImageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device memory for Red Channel
    float *d_R = NULL;
    err = hipMalloc((void **)&d_R, paddedImageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //  Copy image from host memory to device memory 
    printf("Copying image from host memory to device memory.\n");
    err = hipMemcpy(d_B, h_B, paddedImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy blue channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_G, h_G, paddedImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy green channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_R, h_R, paddedImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy red channel image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device memory for output
    float *d_output = NULL;
    err = hipMalloc((void **)&d_output, imageSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory output gradient values
    float *h_output = (float *)malloc(imageSize);

    // Size of image block that will have its gradient calc. in one kernel call
    int blockX = 32, blockY = 32;

    // Size to be allocated for shared memory inside kernel
    int tileX = blockX + padding;
    int tileY = blockY + padding; 
    size_t tileSize = (tileX)*(tileY)*sizeof(float);

    // Specifying execution configuration
    dim3 X(ceil(img.rows/blockX),ceil(img.cols/blockY));
    dim3 Y(tileX/4  ,tileY);

    convolution<<<X, Y, tileSize>>>(d_B, paddedX, paddedY, blockX, blockY, d_output);
   
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch convolution kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output, d_output, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Verify that the resulting image is correct
    Mat featureImage(img.rows, img.cols, CV_8UC1, Scalar(0));
    for (int i = 0; i < img.rows*img.cols; ++i)
    {
        featureImage.at<uchar>(i/img.cols,i%img.cols) = h_output[i];
    }
    imshow("Output Image", featureImage);
    waitKey(0);

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_output);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_G);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array G (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_R);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device array R (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_B);
    free(h_G);
    free(h_R);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

